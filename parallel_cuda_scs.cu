/* Author: zhaojer */

#include <hip/hip_runtime.h>
#include <string>
#include <fstream>

#define ALPHABET_SIZE 26
#define CONVERT_LETTER_TO_IDX(letter) (int(letter) - 97)
#define MIN(a, b) ((a) < (b) ? (a) : (b))

// const char ALPHABET[ALPHABET_SIZE] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z'};
__device__ const char d_ALPHABET[ALPHABET_SIZE] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z'};


__global__ void compute_j_minus_k(int* A, const char* s2, const int m)
{
    // sanity check
    // printf("Block Id: %d, Thread Id: %d\n", blockIdx.x, threadIdx.x);
    // printf("String Y: %s, m = %d\n", s2, m);
    // for (int i = 0; i < ALPHABET_SIZE; ++i) {
    //     printf("%c ", d_ALPHABET[i]);
    //     for (int j = 0; j <= m; ++j) {
    //         printf("%d ", A[i*ALPHABET_SIZE + j]);
    //     }
    //     printf("\n");
    // }

    // calculate index which current thread needs to access
    // threadIdx.x simply corresponds to the row index i
    // note: no need to use blockIdx bc assumption: size of alphabet <= 1024
    int startIdx = threadIdx.x * (m+1);
    // each thread loops an entire "row" in A
    int endIdx = startIdx + m;
    // both indices are inclusive
    // printf("Block Id: %d, Thread Id: %d, Start Index: %d, End Indx: %d\n", blockIdx.x, threadIdx.x, startIdx, endIdx);
    int j = 1;
    for (int idx = startIdx + 1; idx <= endIdx; ++idx) {
        if (s2[j-1] == d_ALPHABET[threadIdx.x])
            A[idx] = j;
        else
            A[idx] = A[idx-1];
        ++j;
    }
}

__global__ void compute_scs_0th_row(int* M, const int m)
{
    // sanity check
    // printf("Block Id: %d, Thread Id: %d\n", blockIdx.x, threadIdx.x);
    // printf("String X: %s, String Y: %s, n = %d, m = %d\n", s1, s2, n, m);
    // for (int i = 0; i < ALPHABET_SIZE; ++i) {
    //     printf("%c ", d_ALPHABET[i]);
    //     for (int j = 0; j <= m; ++j) {
    //         printf("%d ", A[i*ALPHABET_SIZE + j]);
    //     }
    //     printf("\n");
    // }
    // compute the 0th row, base case
    int j_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (j_idx > m)
        return;
    // printf("Block Id: %d, Thread Id: %d, Idx: %d\n", blockIdx.x, threadIdx.x, j_idx);
    M[j_idx] = j_idx;
}

__global__ void compute_scs(int* M, const int* A, const char* s1, const char* s2, const long long i_idx, const long long m)
{
    // sanity check
    // printf("Block Id: %d, Thread Id: %d\n", blockIdx.x, threadIdx.x);
    // printf("String X: %s, String Y: %s, n = %d, m = %d\n", s1, s2, n, m);
    // for (int i = 0; i < ALPHABET_SIZE; ++i) {
    //     printf("%c ", d_ALPHABET[i]);
    //     for (int j = 0; j <= m; ++j) {
    //         printf("%d ", A[i*ALPHABET_SIZE + j]);
    //     }
    //     printf("\n");
    // }
    // find corresponding column index
    const long long j_idx = threadIdx.x + (long long)blockIdx.x * blockDim.x;
    // check for boundaries
    if (j_idx > m)
        return;
    // find index into M when M is flattened
    const long long idx_i_j = i_idx * (m+1) + j_idx;
    // printf("Block Id: %d, Thread Id: %d, Idx: %d\n", blockIdx.x, threadIdx.x, idx_i_j);
    // base case
    if (j_idx == 0) {
        M[idx_i_j] = i_idx;
    }
    else {
        const long long idx_i_minus_1_j = (i_idx-1) * (m+1) + j_idx;
        const int j_minus_k = A[CONVERT_LETTER_TO_IDX(s1[i_idx-1]) * (m+1) + j_idx];
        const int k = j_idx - j_minus_k;
        int M_i_j_minus_1;
        if (j_minus_k == 0)
            M_i_j_minus_1 = i_idx + k - 1;
        else
            M_i_j_minus_1 = M[(i_idx-1) * (m+1) + (j_minus_k-1)] + k;
        // compute current value
        M[idx_i_j] = 1 + MIN(M_i_j_minus_1, M[idx_i_minus_1_j]);
    }
}

//host function, __host__ qualifier assumed by default
int main(int argc, char** argv)
{
    // get input file name from commandline if one is provided
    std::string input_file;
    if (argc == 1) {
        // default input file name
        input_file = "input/input-2000.txt";
    }
    else if (argc == 2) {
        input_file = argv[1];
    }
    else {
        printf("Error: Invalid number of arguments provided\n");
        printf("Usage: ./<program> <input file>\n");
        return 1;
    }
    printf("Input: %s\n", input_file.c_str());
    // 2 input strings
    std::string X = "ozpxennwaelglzwocdybdmpmmcyconwcmlbsaoqcvciidewfiuiljaavcazqnvvbjyvjpmokqwstboa";
    std::string Y = "iyklqkkdhnvwnrjbxkuyltiaqbllgsipqvaihmlozhnmyypxkjwwegyujjhqepfumhfuvqiuzvixtxxgivcobakllrbriimvrrpmjzgjxqisnfy";
    // read input string from file
    std::ifstream fin;
    fin.open(input_file);
    // throw error if the file opening fails
    if (!fin.is_open()) {
        printf("Error opening file: %s\n", input_file.c_str());
        return 1;
    }
    std::getline(fin, X);
    std::getline(fin, Y);
    fin.close();

    // size of strings
    const long long n = X.size();
    const long long m = Y.size();

    // allocate memory for device variables
    char *d_X, *d_Y;
    // memo directly defined on device memory
    int *d_A; // j - k
    int *d_M; // SCS length
    // check if the cuda functions fail using status codes provided by nvcc compiler
    if (hipMalloc(&d_X, sizeof(char) * (n+1)) != hipSuccess) {
        printf("CUDA Error: Could not allocate d_X for string X\n");
        return 1;
    }
    if (hipMalloc(&d_Y, sizeof(char) * (m+1)) != hipSuccess) {
        printf("CUDA Error: Could not allocate d_Y for string Y\n");
        return 1;
    }
    if (hipMalloc(&d_A, sizeof(int) * (m+1) * ALPHABET_SIZE) != hipSuccess) {
        printf("CUDA Error: Could not allocate d_A for memo A\n");
        return 1;
    }
    if (hipMalloc(&d_M, sizeof(int) * (m+1) * (n+1)) != hipSuccess) {
        printf("CUDA Error: Could not allocate d_M for memo M\n");
        return 1;
    }

    // copy memory to device
    if (hipMemcpy(d_X, X.c_str(), sizeof(char) * (n+1), hipMemcpyHostToDevice) != hipSuccess) {
        printf("CUDA Error: Could not copy X into d_X\n");
        return 1;
    }
    if (hipMemcpy(d_Y, Y.c_str(), sizeof(char) * (m+1), hipMemcpyHostToDevice) != hipSuccess) {
        printf("CUDA Error: Could not copy Y into d_Y\n");
        return 1;
    }
    if (hipMemset(d_A, 0, sizeof(int) * (m+1) * ALPHABET_SIZE) != hipSuccess) {
        printf("CUDA Error: Could not set memory of d_A to 0\n");
        return 1;
    }
    if (hipMemset(d_M, 0, sizeof(int) * (m+1) * (n+1)) != hipSuccess) {
        printf("CUDA Error: Could not set memory of d_M to 0\n");
        return 1;
    }

    // declare & create cuda events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // declare block and grid dimensions
    // only use 1 dimension bc of row-wise and col-wise independence
    // for computing j-k or memmo A, assume ALPHABET_SIZE <= 1024
    dim3 blockDimA(ALPHABET_SIZE, 1, 1);
    dim3 gridDimA(1, 1, 1);
    // for computing SCS or memo M
    int num_threads = MIN(1024, m+1);
    dim3 blockDimM(num_threads, 1, 1);
    int num_blocks = std::ceil((m+1) / (double)1024);
    dim3 gridDimM(num_blocks, 1, 1);
    // printf("Sanity Check, Number of Threads %d, Number of Blocks: %d\n", num_threads, num_blocks);

    // record time for start
    hipEventRecord(start);

    // Step 1: compute j - k, i.e. memo A
    compute_j_minus_k<<<gridDimA,blockDimA>>>(d_A, d_Y, m);
    // Step 2: compute SCS length, i.e. memo M
    compute_scs_0th_row<<<gridDimM,blockDimM>>>(d_M, m);
    for (int i = 1; i <= n; ++i) {
        compute_scs<<<gridDimM,blockDimM>>>(d_M, d_A, d_X, d_Y, i, m);
    }

    // record time for stop
    hipEventRecord(stop);
    // blocks CPU execution until the specified event is recorded
    hipEventSynchronize(stop);

    /* ---------------- Copy stuff back from GPU to CPU ---------------- */
    // DEBUG
    // if (cudaMemcpy(A, d_A, sizeof(int) * (m+1) * ALPHABET_SIZE, cudaMemcpyDeviceToHost) != cudaSuccess) {
    //     printf("CUDA Error: Could not copy d_A from device back to A in host\n");
    // }
    // if (cudaMemcpy(M, d_M, sizeof(int) * (m+1) * (n+1), cudaMemcpyDeviceToHost) != cudaSuccess) {
    //     printf("CUDA Error: Could not copy d_M from device back to M in host\n");
    // }
    // for (int i = 0; i < ALPHABET_SIZE; ++i) {
    //     printf("%c ", ALPHABET[i]);
    //     for (int j = 0; j <= m; ++j) {
    //         printf("%d ", A[i][j]);
    //     }
    //     printf("\n");
    // }
    // for (int i = 0; i <= n; ++i) {
    //     for (int j = 0; j <= m; ++j) {
    //         printf("%d ", M[i][j]);
    //     }
    //     printf("\n");
    // }
    // END DEBUG
    int scs_length;
    if (hipMemcpy(&scs_length, &d_M[n * (m+1) + m], sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("CUDA Error: Could not copy final SCS length from device back to host\n");
    }
    /* --------------- Print elapsed time & verification --------------- */
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Execution Time (ms) %f\n", elapsed_time);
    printf("Length of SCS is %d\n", scs_length);

    // clean up
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_A);
    hipFree(d_M);

    return 0;
}