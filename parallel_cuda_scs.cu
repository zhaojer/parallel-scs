/* Author: zhaojer */

#include <hip/hip_runtime.h>
#include <string>
#include <fstream>

#define ALPHABET_SIZE 26
#define CONVERT_LETTER_TO_IDX(letter) (int(letter) - 97)
#define MIN(a, b) ((a) < (b) ? (a) : (b))

static const char ALPHABET[ALPHABET_SIZE] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z'};

// REQUIRES: nums only has 5 numbers
// MODIFIES: nums
// EFFECTS: sort nums up to 3 elements using a bubble sort; median is nums[2]
__device__ void bubleSort(double nums[])
{
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 5 - i - 1; j++) {
            if (nums[j] > nums[j + 1]) {
                double temp = nums[j];
                nums[j] = nums[j+1];
                nums[j+1] = temp;
            }
        }
    }
}

// REQUIRES: curA and nextA correspond to matrix A and have size N * N
//           curA contains values from previous iteration
// MODIFIES: nextA
// EFFECTS: perform (1 iteration of) stencil calculation
//          by reading from curA and writing to nextA
// __global__ void StencilCalculation(double* curA, double* nextA)
// {
//     /*
//     we are using 2-Dimensional grid and 2-Dimensional block for this kernel
//     this structure/setup is automatically partioning the 2D matrix A for us
//     think of each block as taking non-overlapping partition of A of size [blockDim.x][blockDim.y]
//     e.g. if block dims are 16 by 16, then each block is taking partition_of_A[16][16]
//         and all these blocks together span the entire A
//         see diagram I drew on paper
//     */

//     // declare shared memory for each block, speeds up memory access
//     __shared__ double localA[one_dim + 2][one_dim + 2];
//     // doing +2 on each dimension because of ghost cells

//     // calculate colIdx and rowIdx corresponding to this thread in GPU
//     // actual/global column index in 2D array A this thread corresponds to
//     int colIdx = blockIdx.y * blockDim.y + threadIdx.y;
//     // actual/global row index in 2D array A this thread corresponds to
//     int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
//     // this is the index this thread accesses in the flattened array A
//     int idx = rowIdx * N + colIdx;
//     // if this thread is out of bounds of N, then no need to go further
//     if (colIdx >= N || rowIdx >= N) {
//         // this happens bc N does not always evenly divide blockDim
//         return;
//     }
//     // find index of surrounding elements, needed for finding median
//     int downIdx = (rowIdx + 1) * N + colIdx;
//     int upIdx = (rowIdx - 1) * N + colIdx;
//     int rightIdx = rowIdx * N + (colIdx + 1);
//     int leftIdx = rowIdx * N + (colIdx - 1);
//     // local index is now 1-based indexing b/c of ghost cells up/down/left/right
//     int localRowIdx = threadIdx.x + 1;
//     int localColIdx = threadIdx.y + 1;
//     // read from global to shared to speedup later access
//     localA[localRowIdx][localColIdx] = curA[idx];
//     if (localRowIdx == 1 && rowIdx > 0) {
//         localA[localRowIdx - 1][localColIdx] = curA[upIdx];
//     }
//     if (localColIdx == 1 && colIdx > 0) {
//         localA[localRowIdx][localColIdx - 1] = curA[leftIdx];
//     }
//     if (localRowIdx == one_dim && rowIdx < N - 1) {
//         localA[localRowIdx + 1][localColIdx] = curA[downIdx];
//     }
//     if (localColIdx == one_dim && colIdx < N - 1) {
//         localA[localRowIdx][localColIdx + 1] = curA[rightIdx];
//     }
//     // need to make sure all threads have read to shared localA before continue
//     __syncthreads();

//     // need synchronization before moving on to next iteration; this is done in CPU
//     if ((rowIdx == 0) || (rowIdx == N - 1) || (colIdx == 0) || (colIdx == N - 1)) {
//         // A's border remains unchanged throughout the iterations
//         nextA[idx] = localA[localRowIdx][localColIdx];
//     }
//     else {
//         // get all values from shared memory
//         double nums[5] = {localA[localRowIdx][localColIdx],
//                             localA[localRowIdx + 1][localColIdx],
//                             localA[localRowIdx - 1][localColIdx], 
//                             localA[localRowIdx][localColIdx + 1],
//                             localA[localRowIdx][localColIdx - 1]};
//         // need to efficiently find the median out of 5 elements
//         // partial bubble sort is the fast (way faster than insertion)
//         bubleSort(nums);
//         double new_median = nums[2];
//         nextA[idx] = new_median;
//         // need to synchronize all threads here before writing to matrix A
//         // this is done using read/write buffer (indirection), see CPU code
//     }
// }


//host function, __host__ qualifier assumed by default
int main()
{
    // 2 input strings
    std::string X = "ozpxennwaelglzwocdybdmpmmcyconwcmlbsaoqcvciidewfiuiljaavcazqnvvbjyvjpmokqwstboa";
    std::string Y = "iyklqkkdhnvwnrjbxkuyltiaqbllgsipqvaihmlozhnmyypxkjwwegyujjhqepfumhfuvqiuzvixtxxgivcobakllrbriimvrrpmjzgjxqisnfy";
    // size of strings
    const int n = X.size();
    const int m = Y.size();
    // define memo
    int A[ALPHABET_SIZE][m+1]; // j - k
    int M[n+1][m+1];           // SCS length
    // guarantee all entries have been set to 0
    memset(A, 0, sizeof(A));
    memset(M, 0, sizeof(M));
    // DEBUG
    // for (int i = 0; i < ALPHABET_SIZE; ++i) {
    //     printf("%c ", ALPHABET[i]);
    //     for (int j = 0; j <= m; ++j) {
    //         printf("%d ", A[i][j]);
    //     }
    //     printf("\n");
    // }
    // END DEBUG

    // allocate memory for device variables
    char *d_X, *d_Y;
    int *d_A;
    int *d_M;
    // check if the cuda functions fail using status codes provided by nvcc compiler
    if (hipMalloc(&d_X, sizeof(char) * (n+1)) != hipSuccess) {
        printf("CUDA Error: Could not allocate d_X for string X\n");
        return 1;
    }
    if (hipMalloc(&d_Y, sizeof(char) * (m+1)) != hipSuccess) {
        printf("CUDA Error: Could not allocate d_Y for string Y\n");
        return 1;
    }
    if (hipMalloc(&d_A, sizeof(int) * (m+1) * ALPHABET_SIZE) != hipSuccess) {
        printf("CUDA Error: Could not allocate d_A for memo A\n");
        return 1;
    }
    if (hipMalloc(&d_M, sizeof(int) * (m+1) * (n+1)) != hipSuccess) {
        printf("CUDA Error: Could not allocate d_M for memo M\n");
        return 1;
    }

    // copy memory to device
    if (hipMemcpy(d_X, X.c_str(), sizeof(char) * (n+1), hipMemcpyHostToDevice) != hipSuccess) {
        printf("CUDA Error: Could not copy X into d_X\n");
        return 1;
    }
    if (hipMemcpy(d_Y, Y.c_str(), sizeof(char) * (m+1), hipMemcpyHostToDevice) != hipSuccess) {
        printf("CUDA Error: Could not copy Y into d_Y\n");
        return 1;
    }
    if (hipMemcpy(d_A, A, sizeof(int) * (m+1) * ALPHABET_SIZE, hipMemcpyHostToDevice) != hipSuccess) {
        printf("CUDA Error: Could not copy A into d_A\n");
        return 1;
    }
    if (hipMemcpy(d_M, M, sizeof(int) * (m+1) * (n+1), hipMemcpyHostToDevice) != hipSuccess) {
        printf("CUDA Error: Could not copy M into d_M\n");
        return 1;
    }

    // declare & create cuda events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // record time for stop
    hipEventRecord(stop);
    // blocks CPU execution until the specified event is recorded
    hipEventSynchronize(stop);

    /* ---------------- 6. Copy stuff back from GPU to CPU ---------------- */
    if (hipMemcpy(A, d_A, sizeof(int) * (m+1) * ALPHABET_SIZE, hipMemcpyDeviceToHost) != hipSuccess) {
        printf("CUDA Error: Could not copy d_A from device back to A in host\n");
    }
    if (hipMemcpy(M, d_M, sizeof(int) * (m+1) * (n+1), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("CUDA Error: Could not copy d_M from device back to M in host\n");
    }

    // if (cudaMemcpy(&sum, write_buf, sizeof(double), cudaMemcpyDeviceToHost) != cudaSuccess) {
    //     std::cout << "CUDA Error: Could not copy final sum (d_A[0]) back to host" << std::endl;
    //     return 1;
    // }
    // if (cudaMemcpy(&A_n3_n3, d_A_n3_n3, sizeof(double), cudaMemcpyDeviceToHost) != cudaSuccess) {
    //     std::cout << "CUDA Error: Could not copy final d_A[N/3][N/3] back to host" << std::endl;
    //     return 1;
    // }
    // if (cudaMemcpy(&A_19_37, d_A_19_37, sizeof(double), cudaMemcpyDeviceToHost) != cudaSuccess) {
    //     std::cout << "CUDA Error: Could not copy final d_A[19][37] back to host" << std::endl;
    //     return 1;
    // }
    /* --------------- 7. Print elapsed time & verification --------------- */
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);

    // clean up
    hipFree(d_A);
    hipFree(d_M);

    return 0;
}