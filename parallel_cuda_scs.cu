/* Author: zhaojer */

#include <hip/hip_runtime.h>
#include <string>
#include <fstream>

#define ALPHABET_SIZE 26
#define CONVERT_LETTER_TO_IDX(letter) (int(letter) - 97)
#define MIN(a, b) ((a) < (b) ? (a) : (b))

const char ALPHABET[ALPHABET_SIZE] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z'};
__device__ const char d_ALPHABET[ALPHABET_SIZE] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z'};


__global__ void compute_j_minus_k(int* A, const char* s2, const int m)
{
    // sanity check
    // printf("Block Id: %d, Thread Id: %d\n", blockIdx.x, threadIdx.x);
    // printf("String Y: %s, m = %d\n", s2, m);
    // for (int i = 0; i < ALPHABET_SIZE; ++i) {
    //     printf("%c ", d_ALPHABET[i]);
    //     for (int j = 0; j <= m; ++j) {
    //         printf("%d ", A[i*ALPHABET_SIZE + j]);
    //     }
    //     printf("\n");
    // }

    // calculate index which current thread needs to access
    // threadIdx.x simply corresponds to the row index i
    // note: no need to use blockIdx bc assumption: size of alphabet <= 1024
    int startIdx = threadIdx.x * (m+1);
    // each thread loops an entire "row" in A
    int endIdx = startIdx + m;
    // both indices are inclusive
    // printf("Block Id: %d, Thread Id: %d, Start Index: %d, End Indx: %d\n", blockIdx.x, threadIdx.x, startIdx, endIdx);
    int j = 1;
    for (int idx = startIdx + 1; idx <= endIdx; ++idx) {
        if (s2[j-1] == d_ALPHABET[threadIdx.x])
            A[idx] = j;
        else
            A[idx] = A[idx-1];
        ++j;
    }
}

__global__ void compute_scs_0th_row(int* M, const int m)
{
    // sanity check
    // printf("Block Id: %d, Thread Id: %d\n", blockIdx.x, threadIdx.x);
    // printf("String X: %s, String Y: %s, n = %d, m = %d\n", s1, s2, n, m);
    // for (int i = 0; i < ALPHABET_SIZE; ++i) {
    //     printf("%c ", d_ALPHABET[i]);
    //     for (int j = 0; j <= m; ++j) {
    //         printf("%d ", A[i*ALPHABET_SIZE + j]);
    //     }
    //     printf("\n");
    // }
    // compute the 0th row, base case
    int j_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (j_idx > m)
        return;
    // printf("Block Id: %d, Thread Id: %d, Idx: %d\n", blockIdx.x, threadIdx.x, j_idx);
    M[j_idx] = j_idx;
}

__global__ void compute_scs(int* M, const int* A, const char* s1, const char* s2, const int i_idx, const int m)
{
    // sanity check
    // printf("Block Id: %d, Thread Id: %d\n", blockIdx.x, threadIdx.x);
    // printf("String X: %s, String Y: %s, n = %d, m = %d\n", s1, s2, n, m);
    // for (int i = 0; i < ALPHABET_SIZE; ++i) {
    //     printf("%c ", d_ALPHABET[i]);
    //     for (int j = 0; j <= m; ++j) {
    //         printf("%d ", A[i*ALPHABET_SIZE + j]);
    //     }
    //     printf("\n");
    // }
    // find corresponding column index
    const int j_idx = threadIdx.x + blockIdx.x * blockDim.x;
    // check for boundaries
    if (j_idx > m)
        return;
    // find index into M when M is flattened
    const int idx_i_j = i_idx * (m+1) + j_idx;
    // printf("Block Id: %d, Thread Id: %d, Idx: %d\n", blockIdx.x, threadIdx.x, idx_i_j);
    // base case
    if (j_idx == 0) {
        M[idx_i_j] = i_idx;
    }
    else {
        const int idx_i_minus_1_j = (i_idx-1) * (m+1) + j_idx;
        const int j_minus_k = A[CONVERT_LETTER_TO_IDX(s1[i_idx-1]) * (m+1) + j_idx];
        const int k = j_idx - j_minus_k;
        int M_i_j_minus_1;
        if (j_minus_k == 0)
            M_i_j_minus_1 = i_idx + k - 1;
        else
            M_i_j_minus_1 = M[(i_idx-1) * (m+1) + (j_minus_k-1)] + k;
        // compute current value
        M[idx_i_j] = 1 + MIN(M_i_j_minus_1, M[idx_i_minus_1_j]);
    }
}

//host function, __host__ qualifier assumed by default
int main()
{
    // 2 input strings
    std::string X = "ozpxennwaelglzwocdybdmpmmcyconwcmlbsaoqcvciidewfiuiljaavcazqnvvbjyvjpmokqwstboa";
    std::string Y = "iyklqkkdhnvwnrjbxkuyltiaqbllgsipqvaihmlozhnmyypxkjwwegyujjhqepfumhfuvqiuzvixtxxgivcobakllrbriimvrrpmjzgjxqisnfy";
    // size of strings
    const int n = X.size();
    const int m = Y.size();
    // define memo
    int A[ALPHABET_SIZE][m+1]; // j - k
    int M[n+1][m+1];           // SCS length
    // guarantee all entries have been set to 0
    memset(A, 0, sizeof(A));
    memset(M, 0, sizeof(M));

    // allocate memory for device variables
    char *d_X, *d_Y;
    int *d_A;
    int *d_M;
    // check if the cuda functions fail using status codes provided by nvcc compiler
    if (hipMalloc(&d_X, sizeof(char) * (n+1)) != hipSuccess) {
        printf("CUDA Error: Could not allocate d_X for string X\n");
        return 1;
    }
    if (hipMalloc(&d_Y, sizeof(char) * (m+1)) != hipSuccess) {
        printf("CUDA Error: Could not allocate d_Y for string Y\n");
        return 1;
    }
    if (hipMalloc(&d_A, sizeof(int) * (m+1) * ALPHABET_SIZE) != hipSuccess) {
        printf("CUDA Error: Could not allocate d_A for memo A\n");
        return 1;
    }
    if (hipMalloc(&d_M, sizeof(int) * (m+1) * (n+1)) != hipSuccess) {
        printf("CUDA Error: Could not allocate d_M for memo M\n");
        return 1;
    }

    // copy memory to device
    if (hipMemcpy(d_X, X.c_str(), sizeof(char) * (n+1), hipMemcpyHostToDevice) != hipSuccess) {
        printf("CUDA Error: Could not copy X into d_X\n");
        return 1;
    }
    if (hipMemcpy(d_Y, Y.c_str(), sizeof(char) * (m+1), hipMemcpyHostToDevice) != hipSuccess) {
        printf("CUDA Error: Could not copy Y into d_Y\n");
        return 1;
    }
    if (hipMemcpy(d_A, A, sizeof(int) * (m+1) * ALPHABET_SIZE, hipMemcpyHostToDevice) != hipSuccess) {
        printf("CUDA Error: Could not copy A into d_A\n");
        return 1;
    }
    if (hipMemcpy(d_M, M, sizeof(int) * (m+1) * (n+1), hipMemcpyHostToDevice) != hipSuccess) {
        printf("CUDA Error: Could not copy M into d_M\n");
        return 1;
    }

    // declare & create cuda events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // declare block and grid dimensions
    // only use 1 dimension bc of row-wise and col-wise independence
    // for computing j-k or memmo A, assume ALPHABET_SIZE <= 1024
    dim3 blockDimA(ALPHABET_SIZE, 1, 1);
    dim3 gridDimA(1, 1, 1);
    // for computing SCS or memo M
    int num_threads = MIN(1024, m+1);
    dim3 blockDimM(num_threads, 1, 1);
    int num_blocks = std::ceil((m+1) / (double)1024);
    dim3 gridDimM(num_blocks, 1, 1);
    printf("Sanity Check, Number of Threads %d, Number of Blocks: %d\n", num_threads, num_blocks);

    // record time for start
    hipEventRecord(start);

    // Step 1: compute j - k, i.e. memo A
    compute_j_minus_k<<<gridDimA,blockDimA>>>(d_A, d_Y, m);
    // Step 2: compute SCS length, i.e. memo M
    compute_scs_0th_row<<<gridDimM,blockDimM>>>(d_M, m);
    for (int i = 1; i <= n; ++i) {
        compute_scs<<<gridDimM,blockDimM>>>(d_M, d_A, d_X, d_Y, i, m);
    }

    // record time for stop
    hipEventRecord(stop);
    // blocks CPU execution until the specified event is recorded
    hipEventSynchronize(stop);

    /* ---------------- 6. Copy stuff back from GPU to CPU ---------------- */
    if (hipMemcpy(A, d_A, sizeof(int) * (m+1) * ALPHABET_SIZE, hipMemcpyDeviceToHost) != hipSuccess) {
        printf("CUDA Error: Could not copy d_A from device back to A in host\n");
    }
    if (hipMemcpy(M, d_M, sizeof(int) * (m+1) * (n+1), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("CUDA Error: Could not copy d_M from device back to M in host\n");
    }
    // DEBUG
    // for (int i = 0; i < ALPHABET_SIZE; ++i) {
    //     printf("%c ", ALPHABET[i]);
    //     for (int j = 0; j <= m; ++j) {
    //         printf("%d ", A[i][j]);
    //     }
    //     printf("\n");
    // }
    for (int i = 0; i <= n; ++i) {
        for (int j = 0; j <= m; ++j) {
            printf("%d ", M[i][j]);
        }
        printf("\n");
    }
    // END DEBUG

    // if (cudaMemcpy(&sum, write_buf, sizeof(double), cudaMemcpyDeviceToHost) != cudaSuccess) {
    //     std::cout << "CUDA Error: Could not copy final sum (d_A[0]) back to host" << std::endl;
    //     return 1;
    // }
    // if (cudaMemcpy(&A_n3_n3, d_A_n3_n3, sizeof(double), cudaMemcpyDeviceToHost) != cudaSuccess) {
    //     std::cout << "CUDA Error: Could not copy final d_A[N/3][N/3] back to host" << std::endl;
    //     return 1;
    // }
    // if (cudaMemcpy(&A_19_37, d_A_19_37, sizeof(double), cudaMemcpyDeviceToHost) != cudaSuccess) {
    //     std::cout << "CUDA Error: Could not copy final d_A[19][37] back to host" << std::endl;
    //     return 1;
    // }
    /* --------------- 7. Print elapsed time & verification --------------- */
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Elapsed Time (ms) = %f\n", elapsed_time);

    // clean up
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_A);
    hipFree(d_M);

    return 0;
}